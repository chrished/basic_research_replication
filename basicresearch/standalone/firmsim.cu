#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <mex.h>

#include "mersennetwister/MersenneTwister.h"
#include "mersennetwister/MersenneTwister_kernel.cu"

using namespace std;

///////////////////////////////////////////////////////////////////////////////
// Constants
///////////////////////////////////////////////////////////////////////////////
//#define nF 16384
#define nF 32768
//#define nF 65536

#define R_BURN_INIT 10000
#define R_BURN_MED 2000

#define T_PERIODS 1

#define RAND_CHUNKS 5
#define N_QDISTS 3

#define nN 64
#define nM 10

#define nS 8
#define nSpow 3
#define STATE_NOTHING      0
#define STATE_LOSE         1
#define STATE_GAIN_APP     2
#define STATE_GAIN_BAS_1   3
#define STATE_GAIN_BAS_2   4
#define STATE_GAIN_FREE    5
#define STATE_EXPAND       6
#define STATE_EXIT         7

#define BLOCK_SIZE_F 16

///////////////////////////////////////////////////////////////////////////////
// Common host and device function 
///////////////////////////////////////////////////////////////////////////////

//ceil(a / b)
extern "C" int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//floor(a / b)
extern "C" int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
extern "C" int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
extern "C" int iAlignDown(int a, int b){
    return a - a % b;
}

void Check_CUDA_Error(const char *message)
{
   hipError_t error = hipGetLastError();
   if(error!=hipSuccess) {
      fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
      //exit(-1);
   }
}

///////////////////////////////////////////////////////////////////////////////
// Device kernels
///////////////////////////////////////////////////////////////////////////////

// binary search (only powers of 2 sizes)
__device__ inline int dsamp2(double* pbeg, int pow2, double r)
{
  int len = 1<<(pow2-1);
  int pos = len-1;

  double val;
  for (int s = 0; s < (pow2-1); s++) {
    len /= 2;
    val = pbeg[pos];
    pos += (r < val) ? -len : len;
  }

  val = pbeg[pos];
  pos += (r > val) ? 1 : 0;

  return pos;
}

__global__ void fqsim(double* fquals, double qdec, double qmin)
{
  int q = blockDim.x*blockIdx.x + threadIdx.x;
  int f = blockDim.y*blockIdx.y + threadIdx.y;

  if ((f < nF) && (q < nN)) {
    int i = f*nN+q;
    double qval = fquals[i]*qdec;
    fquals[i] = (qval > qmin) ? qval : qmin;
  } 
}

__global__ void fzsim(int* fnprod, int* fmind, int* fage, double* fquals, int* fpinds, int* fexited,
                      double* qdists, double* qbins, double* tvecs, double* fnrand, int nBpow, int nB, int primetime)
{
  int f = blockDim.x*blockIdx.x + threadIdx.x;

  if (f < nF) {
    int n = fnprod[f];
    int m = fmind[f];
    int age = fage[f];

    int exited = 0;
    if (primetime == 1) {
      exited = fexited[f];
    }

    int* pinds = fpinds + nN*f;
    double* quals = fquals + nN*f;

    int nind;
    int qind;
    int mind;

    // requests
    double* qreq1 = NULL;
    double* qreq2 = NULL;

    // randoms
    double r0 = fnrand[f];
    double r1 = fnrand[nF+f];
    double r2 = fnrand[2*nF+f];
    double r3 = fnrand[3*nF+f];
    double r4 = fnrand[4*nF+f];

    // the sampler
    double* tvec = tvecs + (m-1)*(nN+1)*nS + n*nS;
    int s = dsamp2(tvec,nSpow,r0);
    switch (s) {
      case STATE_EXIT:
        n = 0;
        break;
      case STATE_GAIN_APP:
        qreq1 = qdists;
        break;
      case STATE_GAIN_FREE:
        qreq1 = qdists+2*nB;
        break;
      case STATE_GAIN_BAS_1:
        qreq1 = qdists+nB;
        break;
      case STATE_GAIN_BAS_2:
        qreq1 = qdists+nB;
        qreq2 = qdists+nB;
        break;
      case STATE_LOSE:
        if (n > 1) {
          nind = floor(r1*n);

          quals[nind] = quals[n-1];
          pinds[nind] = pinds[n-1];
        }
        n--;
        break;
      case STATE_EXPAND:
        if (m < nM) {
          qind = dsamp2(qdists,nBpow,r1);

          quals[n] = qbins[qind];
          pinds[n] = m;

          m++;
          n++;
        }
        break;
    }

    age++;

    if (n == 0) {
      exited = 1;
      n = 1;
      m = 1;
      age = 0;

      qind = dsamp2(qdists,nBpow,r3);

      quals[0] = qbins[qind];
      pinds[0] = 0;
    }

    if ((qreq1 != NULL) && (n < nN)) {
      qind = dsamp2(qreq1,nBpow,r1);
      mind = floor(r3*m);

      quals[n] = qbins[qind];
      pinds[n] = mind;

      n++;
    }

    if ((qreq2 != NULL) && (n < nN)) {
      qind = dsamp2(qreq2,nBpow,r2);
      mind = floor(r4*m);

      quals[n] = qbins[qind];
      pinds[n] = mind;

      n++;
    }

    fnprod[f] = n;
    fmind[f] = m;
    fage[f] = age;

    if (primetime == 1) {
      fexited[f] = exited;
    }
  }
}

__global__ void calc_qpow(int* fnprod, double* fquals, double* fqpow1, double pow1, double qmin)
{
  int f = blockDim.x*blockIdx.x + threadIdx.x;

  if (f < nF) {
    int n = fnprod[f];
    double* quals = fquals + nN*f;

    double qp1 = 0.0;
    for (int i = 0; i < n; i++) {
      qp1 += powf(quals[i],pow1);
    }

    fqpow1[f] = qp1;
  }
}

__global__ void calc_mpos(int* fnprod, int* fpinds, int* fmpos)
{
  int f = blockDim.x*blockIdx.x + threadIdx.x;

  if (f < nF) {
    int n = fnprod[f];
    int* pinds = fpinds + nN*f;

    int mind;
    int mpres[nM] = {0,0,0,0,0,0,0,0,0,0};
    for (int i = 0; i < n; i++) {
      mind = pinds[i];
      mpres[mind] = 1;
    }

    int npos = 0;
    for (mind = 0; mind < nM; mind++) {
      if (mpres[mind] == 1) npos++;
    }

    fmpos[f] = npos;
  }
}

__global__ void sampn(int* fnprod, int* fmind, int* fage, double* fquals, int* fpinds,
                      double* qdist, double* qbins, double* fnrand, int nBpow, int nB)
{
  int f = blockDim.x*blockIdx.x + threadIdx.x;

  if (f < nF) {
    int* pinds = fpinds + nM*f;
    double* quals = fquals + nN*f;

    int qind = dsamp2(qdist+2*nB,nBpow,fnrand[f]);
    double qval = qbins[qind];

    fnprod[f] = 1;
    fmind[f] = 1;
    fage[f] = 0;
    quals[0] = qval;
    pinds[0] = 0;
  }
}

///////////////////////////////////////////////////////////////////////////////
// persistent memory
///////////////////////////////////////////////////////////////////////////////

// Memory sizes
static int N_PER_RNG = 0;
static int RAND_N = 0;

static int nBpow = 0;
static int nB = 0;

static int R_BURN = 0;
static int R_SIM = 0;
static int R_TOT = 0;
static int R_PER_T = 0;

static int nel_ft = 0;
static int nel_fq = 0;
static int nel_fm = 0;
static int nel_em = 0;

static size_t fqsize = 0;
static size_t fmsize = 0;
static size_t ftsize = 0;
static size_t nrsize = 0;
static size_t qdsize = 0;
static size_t qbsize = 0;
static size_t ttsize = 0;
static size_t emsize = 0;

// Host memory
static double* h_ttable = NULL;

// Device memory
static int* d_fnprod = NULL;
static int* d_fmind = NULL;
static int* d_fpinds = NULL;
static int* d_fage = NULL;
static double* d_fqpow1 = NULL;
static double* d_fquals = NULL;
static int* d_fexited = NULL;

static int* d_fmpos = NULL;
static double* d_fnrand = NULL;
static double* d_qdists = NULL;
static double* d_qbins = NULL;
static double* d_ttable = NULL;

// Flags
static int initialized = 0;

// Mersenne info
const char *dat_path = "standalone/mersennetwister/data/MersenneTwister.dat";
static unsigned int SEED = 191871;

//MTRand* mt = NULL;

///////////////////////////////////////////////////////////////////////////////
// initialization/deinitialization code
///////////////////////////////////////////////////////////////////////////////

void cleanup() {
  if (initialized == 1) {
    printf("Deinitializing firmsim_cu.\n");

    // Free device memory
    hipFree(d_fnprod);
    hipFree(d_fmind);
    hipFree(d_fpinds);
    hipFree(d_fage);
    hipFree(d_fqpow1);
    hipFree(d_fquals);
    hipFree(d_fexited);

    hipFree(d_fmpos);
    hipFree(d_fnrand);
    hipFree(d_qdists);
    hipFree(d_qbins);
    hipFree(d_ttable);

    // Free host memory
    free(h_ttable);

    // Set initialized flag
    initialized = 0;
  }
}

void initialize(int nBpow_in, int R_PER_T_IN) {
  if (initialized == 0) {
    printf("Initializing firmsim_cu.\n");

    //mt = new MTRand(SEED);

    nBpow = nBpow_in;
    nB = 1<<nBpow;

    R_PER_T = R_PER_T_IN;
    R_SIM = T_PERIODS*R_PER_T;

    // Initialize MersenneTwister
    N_PER_RNG = iAlignUp(iDivUp(nF,MT_RNG_COUNT),2);
    RAND_N = MT_RNG_COUNT*N_PER_RNG;
    loadMTGPU(dat_path);
    seedMTGPU(SEED);

    // element counts
    nel_ft = nF;
    nel_fq = nN*nF;
    nel_fm = nN*nF;
    nel_em = nF;

    // Set up memory sizes
    fqsize = sizeof(double)*nel_fq;
    fmsize = sizeof(double)*nel_fm;
    ftsize = sizeof(int)*nel_ft;
    nrsize = sizeof(double)*RAND_CHUNKS*RAND_N;
    qbsize = sizeof(double)*nB;
    qdsize = sizeof(double)*N_QDISTS*nB;
    emsize = sizeof(double)*nel_em;
    ttsize = sizeof(double)*nM*(nN+1)*nS;

    // Allocate host memory
    h_ttable = (double*)malloc(ttsize);

    // Allocate device memory
    hipMalloc((void**)&d_fnprod,ftsize);
    hipMalloc((void**)&d_fmind,ftsize);
    hipMalloc((void**)&d_fpinds,fmsize);
    hipMalloc((void**)&d_fage,ftsize);
    hipMalloc((void**)&d_fqpow1,emsize);
    hipMalloc((void**)&d_fquals,fqsize);
    hipMalloc((void**)&d_fexited,ftsize);

    hipMalloc((void**)&d_fmpos,ftsize);
    hipMalloc((void**)&d_fnrand,nrsize);
    hipMalloc((void**)&d_qdists,qdsize);
    hipMalloc((void**)&d_qbins,qbsize);
    hipMalloc((void**)&d_ttable,ttsize);

    Check_CUDA_Error("Failed hipMalloc.");

    // Set initialized flag
    mexAtExit(cleanup);
    initialized = 1;
  }
}

///////////////////////////////////////////////////////////////////////////////
// MEX code
///////////////////////////////////////////////////////////////////////////////
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
  // read in arguments
  if (nrhs != 15) {
    printf("Too few input arguments: nrhs = %i.\n",nrhs);
    return;
  }

  if (nlhs != 11) {
    printf("Too few output arguments: nlhs = %i.\n",nlhs);
    return;
  }

  // Handle matlab data
  int nBpow_in = mxGetScalar(prhs[0]);
  double* m_qbins = (double*)mxGetData(prhs[1]);
  double* m_qdists = (double*)mxGetData(prhs[2]);
  double* m_xa = (double*)mxGetData(prhs[3]);
  double* m_xb = (double*)mxGetData(prhs[4]);
  double xe = mxGetScalar(prhs[5]);
  double* m_rho = (double*)mxGetData(prhs[6]);
  double* m_epr = (double*)mxGetData(prhs[7]);
  double tau = mxGetScalar(prhs[8]);
  double g = mxGetScalar(prhs[9]);
  double r = mxGetScalar(prhs[10]);
  double eps = mxGetScalar(prhs[11]);
  double kappa = mxGetScalar(prhs[12]);
  double qmin = mxGetScalar(prhs[13]);
  int* m_r_per_t = (int*)mxGetData(prhs[14]);
  int R_PER_T_IN = *m_r_per_t;

  int qbN = mxGetN(prhs[1]);
  int qbM = mxGetM(prhs[1]);
  int qbL = max(qbN,qbM);
  if (qbL != (1<<nBpow_in)) {
    printf("nBpow wrong.\n");
    return;
  }

  if ((initialized == 1) && ((nBpow_in != nBpow) || (R_PER_T_IN != R_PER_T))) cleanup();

  // Initialize
  if (initialized == 0) initialize(nBpow_in,R_PER_T_IN);

  R_BURN = R_BURN_INIT;
  R_TOT = R_BURN+R_SIM;

  int gen_init = 1;
  seedMTGPU(SEED);
  //seedMTGPU(mt->randInt());

  double delt = 1.0/R_PER_T;
  double qdec = 1.0/(1.0+delt*g);

  // Make transition table
  double* svec;
  double ssum;
  double xat;
  double xbt;
  double xet;
  double rhot;
  double eprt;
  for (int m = 0; m < nM; m++) {
    xat = m_xa[m];
    xbt = m_xb[m];
    xet = xe;
    rhot = m_rho[m];
    eprt = m_epr[m];

    for (int n = 0; n < nN+1; n++) {
      svec = h_ttable + m*(nN+1)*nS + n*nS;

      ssum = 1.0;
      svec[STATE_EXIT] = ssum;
      ssum -= delt*(kappa);
      svec[STATE_EXPAND] = ssum;
      ssum -= delt*(xet*eprt);
      svec[STATE_GAIN_FREE] = ssum;
      ssum -= delt*(n*kappa);
      svec[STATE_GAIN_BAS_2] = ssum;
      ssum -= delt*(n*xbt*rhot);
      svec[STATE_GAIN_BAS_1] = ssum;
      ssum -= delt*(n*xbt*(1.0-rhot));
      svec[STATE_GAIN_APP] = ssum;
      ssum -= delt*(n*xat+xet*(1.0-eprt));
      svec[STATE_LOSE] = ssum;
      ssum -= delt*(n*tau);
      svec[STATE_NOTHING] = ssum;
    }
  }

  if (ssum <= 0.0) printf("delt too large.");

  // powers
  double pow1 = eps-1.0;

  // output arrays
  mxArray* m_nprod = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);
  mxArray* m_mind = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);
  mxArray* m_age = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);
  mxArray* m_mpos = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);
  mxArray* m_qpow1 = mxCreateDoubleMatrix(nF,1,mxREAL);
  mxArray* m_nprod_zero = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);
  mxArray* m_mind_zero = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);
  mxArray* m_age_zero = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);
  mxArray* m_mpos_zero = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);
  mxArray* m_qpow1_zero = mxCreateDoubleMatrix(nF,1,mxREAL);
  mxArray* m_exited = mxCreateNumericMatrix(nF,1,mxINT32_CLASS,mxREAL);

  int* h_nprod = (int*)mxGetData(m_nprod);
  int* h_mind = (int*)mxGetData(m_mind);
  int* h_age = (int*)mxGetData(m_age);
  int* h_mpos = (int*)mxGetData(m_mpos);
  double* h_qpow1 = (double*)mxGetData(m_qpow1);
  int* h_nprod_zero = (int*)mxGetData(m_nprod_zero);
  int* h_mind_zero = (int*)mxGetData(m_mind_zero);
  int* h_age_zero = (int*)mxGetData(m_age_zero);
  int* h_mpos_zero = (int*)mxGetData(m_mpos_zero);
  double* h_qpow1_zero = (double*)mxGetData(m_qpow1_zero);
  int* h_exited = (int*)mxGetData(m_exited);

  // Block size info
  int block_size_q = 32;
  int block_size_f = 16;

  int n_blocks_q = iDivUp(nN,block_size_q);
  int n_blocks_f = iDivUp(nF,block_size_f);

  dim3 block_size(block_size_q,block_size_f);
  dim3 n_blocks(n_blocks_q,n_blocks_f);

  hipMemcpy(d_qdists,m_qdists,qdsize,hipMemcpyHostToDevice);
  hipMemcpy(d_qbins,m_qbins,qbsize,hipMemcpyHostToDevice);
  hipMemcpy(d_ttable,h_ttable,ttsize,hipMemcpyHostToDevice);

  // Firm inital sizes
  if (gen_init == 1) {
    RandomGPU<<<32,128>>>(d_fnrand,N_PER_RNG);
    Check_CUDA_Error("Failed RandomGPU.");

    sampn<<<n_blocks_f,block_size_f>>>(d_fnprod,d_fmind,d_fage,d_fquals,d_fpinds,d_qdists,d_qbins,d_fnrand,nBpow,nB);
    Check_CUDA_Error("Failed sampn.");
  }

  int primet = 0;
  for (int rep = 0; rep < R_TOT; rep++) {
    if (rep == R_BURN) {
      primet = 1;

      calc_qpow<<<n_blocks_f,block_size_f>>>(d_fnprod,d_fquals,d_fqpow1,pow1,qmin);
      Check_CUDA_Error("Failed calc_qpow.");

      calc_mpos<<<n_blocks_f,block_size_f>>>(d_fnprod,d_fpinds,d_fmpos);
      Check_CUDA_Error("Failed calc_mpos.");

      hipMemcpy(h_nprod_zero,d_fnprod,ftsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_mind_zero,d_fmind,ftsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_age_zero,d_fage,ftsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_mpos_zero,d_fmpos,ftsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_qpow1_zero,d_fqpow1,emsize,hipMemcpyDeviceToHost);
      hipMemset((void*)d_fexited,0,ftsize);
    }

    RandomGPU<<<32,128>>>(d_fnrand,RAND_CHUNKS*N_PER_RNG);
    Check_CUDA_Error("Failed RandomGPU.");

    fzsim<<<n_blocks_f,block_size_f>>>(d_fnprod,d_fmind,d_fage,d_fquals,d_fpinds,d_fexited,d_qdists,d_qbins,d_ttable,d_fnrand,nBpow,nB,primet);
    Check_CUDA_Error("Failed fzsim.");

    fqsim<<<n_blocks,block_size>>>(d_fquals,qdec,qmin);
    Check_CUDA_Error("Failed fqsim.");

    if (rep == R_TOT-1) {
      calc_qpow<<<n_blocks_f,block_size_f>>>(d_fnprod,d_fquals,d_fqpow1,pow1,qmin);
      Check_CUDA_Error("Failed calc_qpow.");

      calc_mpos<<<n_blocks_f,block_size_f>>>(d_fnprod,d_fpinds,d_fmpos);
      Check_CUDA_Error("Failed calc_mpos.");

      hipMemcpy(h_nprod,d_fnprod,ftsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_mind,d_fmind,ftsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_age,d_fage,ftsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_mpos,d_fmpos,ftsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_qpow1,d_fqpow1,emsize,hipMemcpyDeviceToHost);
      hipMemcpy(h_exited,d_fexited,ftsize,hipMemcpyDeviceToHost);
    }
  }

  plhs[0] = m_nprod;
  plhs[1] = m_mind;
  plhs[2] = m_age;
  plhs[3] = m_mpos;
  plhs[4] = m_qpow1;
  plhs[5] = m_nprod_zero;
  plhs[6] = m_mind_zero;
  plhs[7] = m_age_zero;
  plhs[8] = m_mpos_zero;
  plhs[9] = m_qpow1_zero;
  plhs[10] = m_exited;
}

