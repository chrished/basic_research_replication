#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <mex.h>

using namespace std;

const int nM = 10;

///////////////////////////////////////////////////////////////////////////////
// Utility functions
///////////////////////////////////////////////////////////////////////////////

/*
inline int topBlock(int nel, int bs)
{
  return (nel+(bs-nel%bs))/bs;
}
*/

int topBlock(int nel, int bs)
{
  return ((nel%bs) != 0) ? (nel/bs+1) : (nel/bs);
}

void checkCudaError(hipError_t err)
{
  if(!err) return;

  printf("%s\n", hipGetErrorString(err));

  exit(EXIT_FAILURE);
}

///////////////////////////////////////////////////////////////////////////////
// Device kernels
///////////////////////////////////////////////////////////////////////////////

__global__ void spmv(double* vin, double* vout, double pout, double pdown, 
                     double* pup1, double* pup2, double* mup0, double* mup1, int nN)
{
  volatile __shared__ double vsum[nM];
  volatile __shared__ double l_pup1[nM];
  volatile __shared__ double l_pup2[nM];
  volatile __shared__ double l_mup0[nM];
  volatile __shared__ double l_mup1[nM];

  int tm = threadIdx.x;
  int tn = threadIdx.y;

  int m = blockDim.x*blockIdx.x + tm;
  int n = blockDim.y*blockIdx.y + tn;

  if ((m < nM) && (n < nN)) {
    if (tn == 0) {
      l_pup1[m] = pup1[m];
      l_pup2[m] = pup2[m];
      l_mup0[m] = mup0[m];
      l_mup1[m] = mup1[m];
    }
    __syncthreads();

    double vtmp = 0.0;
    int i = m*nN+n;

    // stay
    double pstay = 1.0-pout-(n+1)*pdown;
    if (n<nN-2) pstay -= (n+1)*l_pup2[m];
    if (n<nN-1) pstay -= (n+1)*l_pup1[m]+l_mup0[m];
    if ((m<nM-1)&&(n<nN-1)) pstay -= l_mup1[m];
    vtmp += vin[i]*pstay;

    // transitions
    if (n<nN-1) vtmp += vin[i+1]*(n+2)*pdown;
    if (n>0) vtmp += vin[i-1]*(n*l_pup1[m]+l_mup0[m]);
    if (n>1) vtmp += vin[i-2]*(n-1)*l_pup2[m];
    if ((m>0)&&(n>0)) vtmp += vin[i-nN-1]*l_mup1[m-1];

    // pesky first row, just do it
    if (n == 0) {
      vsum[m] = vin[m*nN];
      if (m < 5)  { vsum[m] += vsum[m+5]; }
      if (m < 2)  { vsum[m] += vsum[m+2]; }
      if (m == 0) { vtmp += pout + (vsum[0] + vsum[1] + vsum[4])*pdown; }
    }

    vout[i] = vtmp;
  }
}

__global__ void dinit(double* ev, double norm, int nN)
{
  int m = blockDim.x*blockIdx.x + threadIdx.x;
  int n = blockDim.y*blockIdx.y + threadIdx.y;

  if ((m < nM) && (n < nN)) {
    int i = m*nN+n;
    ev[i] = norm/double((n+1)*(m+1));
  }
}

__global__ void vec_diff(double* vin, double* vout, int N)
{
  int i = blockDim.x*blockIdx.x + threadIdx.x;

  if (i < N) {
    vout[i] -= vin[i];
  }
}

__global__ void vec_max(double* vin, int N)
{
  __shared__ double sdata[512];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  sdata[tid] = (i < N) ? vin[i] : 0.0;
  __syncthreads();

  // do reduction in shared mem
  if (tid < 256) { sdata[tid] = fmaxf(sdata[tid], sdata[tid + 256]); } __syncthreads();
  if (tid < 128) { sdata[tid] = fmaxf(sdata[tid], sdata[tid + 128]); } __syncthreads();
  if (tid <  64) { sdata[tid] = fmaxf(sdata[tid], sdata[tid +  64]); } __syncthreads();
    
  if (tid < 32)
  {
      sdata[tid] = fmaxf(sdata[tid], sdata[tid + 32]);
      sdata[tid] = fmaxf(sdata[tid], sdata[tid + 16]);
      sdata[tid] = fmaxf(sdata[tid], sdata[tid +  8]);
      sdata[tid] = fmaxf(sdata[tid], sdata[tid +  4]);
      sdata[tid] = fmaxf(sdata[tid], sdata[tid +  2]);
      sdata[tid] = fmaxf(sdata[tid], sdata[tid +  1]);
  }

  // write result for this block to global mem 
  if (tid == 0) vin[blockIdx.x] = sdata[0];
}

///////////////////////////////////////////////////////////////////////////////
// persistent memory
///////////////////////////////////////////////////////////////////////////////

static double* d_vold = NULL;
static double* d_vnew = NULL;
static double* d_pup1 = NULL;
static double* d_pup2 = NULL;
static double* d_mup0 = NULL;
static double* d_mup1 = NULL;
static int initialized = 0;
static int nN = 0;
static double norm = 0.0;

void cleanup(void) {
  printf("Deinitializing eigsim.\n");
  if (initialized == 1) {
    hipFree(d_vold);
    hipFree(d_vnew);
    hipFree(d_pup1);
    hipFree(d_pup2);
    hipFree(d_mup0);
    hipFree(d_mup1);
    
    initialized = 0;
  }
}

///////////////////////////////////////////////////////////////////////////////
// main code
///////////////////////////////////////////////////////////////////////////////
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
  int n,m;

  // read in params
  if (nrhs != 7) {
    printf("Too few arguments.\n");
    return;
  }

  double* h_pup1 = (double*)mxGetData(prhs[0]);
  double* h_pup2 = (double*)mxGetData(prhs[1]);
  double* h_mup0 = (double*)mxGetData(prhs[2]);
  double* h_mup1 = (double*)mxGetData(prhs[3]);
  double h_pdown = mxGetScalar(prhs[4]);
  double h_pout = mxGetScalar(prhs[5]);
  int nNin = mxGetScalar(prhs[6]);

  if ((initialized == 1) && (nNin != nN)) cleanup();
  nN = nNin;

  int nTM = nM*nN;
  size_t msize = sizeof(double)*nM;
  size_t evsize = sizeof(double)*nTM;

  // Create matlab output array
  mxArray* m_ev = mxCreateDoubleMatrix(1,nTM,mxREAL);
  double* h_ev =  (double*)mxGetData(m_ev);
  plhs[0] = m_ev;

  hipError_t err;
  double* d_vtmp;

  dim3 threadsPerBlock(16,16);
  dim3 numBlocks(topBlock(nM,threadsPerBlock.x),topBlock(nN,threadsPerBlock.y));

  dim3 tpb_diff(64,1);
  dim3 nbl_diff(topBlock(nTM,tpb_diff.x),1);

  dim3 tpb_max(512,1);
  dim3 nbl_max(topBlock(nTM,tpb_max.x),1);
  int nred = nbl_max.x;

  hipblasStatus_t stat;

  if (initialized == 0) {
    printf("Initializing eigsim.\n");

    // Initialize CUBLAS
    cublasInit();
    stat = cublasGetError();
    if (stat != HIPBLAS_STATUS_SUCCESS) printf("Error in cublasInit! code = %i\n",stat);

    double sumn = 0.0;
    for (n = 0; n < nN; n++) {
      sumn += 1.0/double(n+1);
    }
    double summ = 0.0;
    for (m = 0; m < nM; m++) {
      summ += 1.0/double(m+1);
    }
    sumn = 1.0/sumn;
    summ = 1.0/summ;
    norm = sumn*summ;

    err = hipMalloc((void**)&d_vold,evsize);
    checkCudaError(err);
    err = hipMalloc((void**)&d_vnew,evsize);
    checkCudaError(err);
    err = hipMalloc((void**)&d_pup1,msize);
    checkCudaError(err);
    err = hipMalloc((void**)&d_pup2,msize);
    checkCudaError(err);
    err = hipMalloc((void**)&d_mup0,msize);
    checkCudaError(err);
    err = hipMalloc((void**)&d_mup1,msize);
    checkCudaError(err);

    dinit<<<numBlocks,threadsPerBlock>>>(d_vold,norm,nN);

    mexAtExit(cleanup);
    initialized = 1;
  } else {
    d_vtmp = d_vold;
    d_vold = d_vnew;
    d_vnew = d_vtmp;
  }

  double* h_vold = (double*)malloc(evsize);
  double* h_vnew = (double*)malloc(evsize);
  double* sum_red = (double*)malloc(nred*sizeof(double));

  err = hipMemcpy(d_pup1,h_pup1,msize,hipMemcpyHostToDevice);
  checkCudaError(err);
  err = hipMemcpy(d_pup2,h_pup2,msize,hipMemcpyHostToDevice);
  checkCudaError(err);
  err = hipMemcpy(d_mup0,h_mup0,msize,hipMemcpyHostToDevice);
  checkCudaError(err);
  err = hipMemcpy(d_mup1,h_mup1,msize,hipMemcpyHostToDevice);
  checkCudaError(err);

  int nchecks = 0;
  int maxiter = nN*2000+1;
  double diff;
  int t;
  for (t = 1; t < maxiter; t++) {
    spmv<<<numBlocks,threadsPerBlock>>>(d_vold,d_vnew,h_pout,h_pdown,d_pup1,d_pup2,d_mup0,d_mup1,nN);

    if (t%5000 == 0) {
      vec_diff<<<nbl_diff,tpb_diff>>>(d_vnew,d_vold,nTM);
      vec_max<<<nbl_max,tpb_max>>>(d_vold,nTM);
      hipMemcpy(sum_red,d_vold,nred*sizeof(double),hipMemcpyDeviceToHost);

      diff = 0.0;
      for (int i = 0; i < nred; i++) {
        diff = max(diff,sum_red[i]);
      }

      nchecks++;
      if (abs(diff) < 1e-12) {
        break;
      }
    }

    d_vtmp = d_vold;
    d_vold = d_vnew;
    d_vnew = d_vtmp;
  }

  //printf("t = %i, nchecks = %i\n",t,nchecks);
  if (t == maxiter) {
    printf("Hit max iterations! diff = %15.12f\n",diff);
  }

  err = hipMemcpy(h_ev,d_vnew,evsize,hipMemcpyDeviceToHost);
  checkCudaError(err);

  /*
  for (m = 0; m < nM; m++) {
    for (n = 0; n < nN; n++) {
      i = m*nN+n;
      sum1 += (n+1)*h_ev[i];
    }
  }
  */

  /*
  double sum1 = 0.0;
  for (int i = 0; i < nTM; i++) {
    sum1 += h_ev[i];
  }
  printf("sum1 = %12.10f\n\n",sum1);
  */

  /*
  for (int tmi = 0; tmi < 10; tmi++) {
    printf("%12.10f\n",h_ev[tmi]);
  }
  printf("\n");
  */
}

